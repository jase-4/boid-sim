#include "boid_sim.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <glm/glm.hpp>


extern "C" void updateBoidsCUDA(glm::vec3* positions, glm::vec3* velocities, int count) {
    glm::vec3* d_positions;
    glm::vec3* d_velocities;

    size_t size = count * sizeof(glm::vec3);
    hipMalloc(&d_positions, size);
    hipMalloc(&d_velocities, size);

    hipMemcpy(d_positions, positions, size, hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, velocities, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (count + threadsPerBlock - 1) / threadsPerBlock;
    updateBoidsKernel<<<blocks, threadsPerBlock>>>(d_positions, d_velocities, count);
    hipDeviceSynchronize();

    hipMemcpy(positions, d_positions, size, hipMemcpyDeviceToHost);
    hipMemcpy(velocities, d_velocities, size, hipMemcpyDeviceToHost);

    hipFree(d_positions);
    hipFree(d_velocities);
}
